#include "hip/hip_runtime.h"
#include "kernel.h"

using namespace std;

#ifndef KERNEL
	#define KERNEL
	Particle *bufferIN, *bufferOUT;
	vector<Particle> &outParticles = vector<Particle>();
	auto dataSize = sizeof(Particle) * PARTICLE_COUNT;
#endif // !KERNEL


// Calculate the forces applying to the particles
__global__ void calcForce(const Particle *in, Particle *out) {
	// Get the thread's unique ID  - (blockIDX * blockDIM) + threadIDX
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	Particle other; // Reference to another particle
	float vel[3] = { 0.0f, 0.0f, 0.0f };
	for (int j = 0; j < PARTICLE_COUNT; j++) {
		other = in[j];
		// Don't calculate against myself
		if (idx == j)
			continue;
		// Calculate the distance between the two particles
		float distVec[3] = {
			other.pos[0] - in[idx].pos[0],
			other.pos[1] - in[idx].pos[1],
			other.pos[2] - in[idx].pos[2]
		};
		// Dot product + softening
		float sqrDist = (distVec[0] * distVec[0] + distVec[1] * distVec[1] + distVec[2] * distVec[2]) + EPS;
		if (sqrDist > 0.1f) {
			float invDist3 = pow(1.0f / sqrtf(sqrDist), 3);
			vel[0] += distVec[0] * invDist3;
			vel[1] += distVec[1] * invDist3;
			vel[2] += distVec[2] * invDist3;
		}
	}
	
	// Update this particle
	out[idx].velocity[0] = in[idx].velocity[0] + PHYSICS_TIME * vel[0] * DAMPENING;
	out[idx].velocity[1] = in[idx].velocity[1] + PHYSICS_TIME * vel[1] * DAMPENING;
	out[idx].velocity[2] = in[idx].velocity[2] + PHYSICS_TIME * vel[2] * DAMPENING;
	out[idx].pos[0] = in[idx].pos[0] + out[idx].velocity[0];
	out[idx].pos[1] = in[idx].pos[1] + out[idx].velocity[1];
	out[idx].pos[2] = in[idx].pos[2] + out[idx].velocity[2];

	// Clamp to bounds
	out[idx].pos[0] = min(max(out[idx].pos[0], -SIM_WIDTH / 2.0f), SIM_WIDTH / 2.0f);
	out[idx].pos[1] = min(max(out[idx].pos[1], -SIM_HEIGHT / 2.0f), SIM_HEIGHT / 2.0f);
	out[idx].pos[2] = min(max(out[idx].pos[2], -SIM_DEPTH / 2.0f), SIM_DEPTH / 2.0f);
}

// Swap the input and output buffers (saves passing data to the GPU every frame)
void swapBuffers() {
	Particle *tempBuffer = bufferIN;
	bufferIN = bufferOUT;
	bufferOUT = tempBuffer;	
}

// Update the particles on the gpu and store them in the passed in vector
void updateParticlesCUDA(const vector<Particle> &particles) {

	calcForce<<<PARTICLE_COUNT / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(bufferIN, bufferOUT);
	hipDeviceSynchronize();
	hipMemcpy((void*)&particles[0], bufferOUT, dataSize, hipMemcpyDeviceToHost);
	
	// Swap the in and out buffers
	swapBuffers();
}

void cudaInfo() {
	// Get CUDA device
	int device;
	hipGetDevice(&device);

	// Get CUDA device
	hipDeviceProp_t properites;
	hipGetDeviceProperties(&properites, device);

	// Display properties
	cout << "|-------------------------------" << endl;
	cout << "|Name: " << properites.name << endl;
	cout << "|CUDA Capability: " << properites.major << "." << properites.minor << endl;
	cout << "|Cores: " << properites.multiProcessorCount << endl;
	cout << "|Memory: " << properites.totalGlobalMem / (1024 * 1024) << "MB" << endl;
	cout << "|Clock freq: " << properites.clockRate / 1000 << "MHz" << endl;
	cout << "|-------------------------------" << endl;
}

void setUpCUDA(const vector<Particle> &particles) {
	cudaInfo();
	hipMalloc((void**)&bufferIN, dataSize);
	hipMalloc((void**)&bufferOUT, dataSize);
	hipMemcpy(bufferIN, &particles.at(0), dataSize, hipMemcpyHostToDevice);
}

// Delete the buffers
void endCUDA() {
	hipFree(bufferIN);
	hipFree(bufferOUT);
}